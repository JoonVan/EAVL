#include "hip/hip_runtime.h"
#include "eavlVolumeRendererMutator.h"
#include "eavlException.h"
#include "eavlExecutor.h"
#include "RT/eavlRTUtil.h"
#include "eavlMapOp.h"
#include "eavlFilter.h"
#include "eavlTimer.h" 
#include "RT/SplitBVH.h"

#define USE_TEXTURE_MEM
#define END_FLAG    -1000000000
#define INFINITE    1000000
#define EPSILON     0.001f

#ifndef HAVE_CUDA
template<class T> class texture {};
struct float4
{
    float x,y,z,w;
};
#endif


/* Triangle textures */
texture<float4> tet_bvh_in_tref;            /* BVH inner nodes */
texture<float4> tet_verts_tref;              /* vert+ scalar data */
texture<float>  tet_bvh_lf_tref;            /* BVH leaf nodes */
texture<float4>  color_map_tref;



template<class T>
class eavlConstArrayV2
{
  public:
    T *host;
    T *device;
  public:
    eavlConstArrayV2(T *from, int N, texture<T> &g_textureRef)
    {
        host = from;

#ifdef HAVE_CUDA
        int nbytes = N * sizeof(T);
        hipMalloc((void**)&device, nbytes);
        CUDA_CHECK_ERROR();
        hipMemcpy(device, &(host[0]),
                   nbytes, hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();
#ifdef USE_TEXTURE_MEM
        hipBindTexture(0, g_textureRef,device,nbytes);
        CUDA_CHECK_ERROR();
#endif

#endif
    }
    ~eavlConstArrayV2()
    {
        

#ifdef HAVE_CUDA
        //hipFree(device);
        //CUDA_CHECK_ERROR();
#endif


    }
#ifdef __CUDA_ARCH__
#ifdef USE_TEXTURE_MEM
    EAVL_DEVICEONLY  const T getValue(texture<T> g_textureRef, int index) const
    {
        return tex1Dfetch(g_textureRef, index);
    }
    EAVL_HOSTONLY  void unbind(texture<T> g_textureRef)
    {
        hipUnbindTexture(g_textureRef);
        CUDA_CHECK_ERROR();
    }
#else
    EAVL_DEVICEONLY const T &getValue(texture<T> g_textureRef, int index) const
    {
        return device[index];
    }
    EAVL_HOSTONLY  void unbind(texture<T> g_textureRef)
    {
        //do nothing
    }
#endif
#else
    EAVL_HOSTONLY const T &getValue(texture<T> g_textureRef, int index) const
    {
        return host[index];
    }
    EAVL_HOSTONLY  void unbind(texture<T> g_textureRef)
    {
        //do nothing
    }
#endif

};


eavlConstArrayV2<float4>* tet_bvh_in_array;
eavlConstArrayV2<float4>* tet_verts_array;
eavlConstArrayV2<float>*  tet_bvh_lf_array;
eavlConstArrayV2<float4>*  color_map_array;

eavlVolumeRendererMutator::eavlVolumeRendererMutator()
{
	height = 1080;
	width  = 1920;
	size = height * width;
	camera.position.x = 10;
	camera.position.y = 0;
	camera.position.z = 0;
	camera.lookat.x = 0;
	camera.lookat.y = 0;
	camera.lookat.z = 0;
	camera.up.x = 0;
	camera.up.y = 1;
	camera.up.z = 0;
	camera.fovx = 45;
	camera.fovy = 30;
	camera.zoom = 1;

	rayOriginX = NULL;
    rayOriginY = NULL;
    rayOriginZ = NULL;
    rayDirX = NULL;
    rayDirY = NULL;
    rayDirZ = NULL;
    indexes = NULL;
    mortonIndexes = NULL;
    tempFloat = NULL;
    r = NULL;
    g = NULL;
    b = NULL;
    a = NULL;

    frameBuffer = NULL;
    geomDirty = true;
    sizeDirty = true;
    verbose = true;

    tet_verts_raw   = NULL;
    tet_bvh_in_raw  = NULL;     
    tet_bvh_lf_raw  = NULL;
    color_map_raw   = NULL;

    tet_bvh_in_array = NULL;
    tet_verts_array  = NULL;
    tet_bvh_lf_array = NULL;
    color_map_array = NULL;
    numTets = 0;
    setDefaultColorMap();
    gpu = true;

    redIndexer   = new eavlArrayIndexer(4,0);
    greenIndexer = new eavlArrayIndexer(4,1);
    blueIndexer  = new eavlArrayIndexer(4,2);
    alphaIndexer = new eavlArrayIndexer(4,3);
}

struct Sample
{
    Sample()
    {
        open = 0;
    }

    int open;
    float d1;
    float d2;
    float s1;
    float s2; 
};

#define CACHE_MAX_SIZE 6
EAVL_HOSTDEVICE eavlVector4 getIntersectionTet(const eavlVector3 rayDir, const eavlVector3 rayOrigin, const eavlConstArrayV2<float4> &bvh,const eavlConstArrayV2<float> &tet_bvh_lf_raw,eavlConstArrayV2<float4> &verts,const float &maxDistance, float &distance, float sampleDelta, eavlConstArrayV2<float4> &cmap, int cmapSize)
{
    cout<<"New ray --------------------------------------------------------"<<endl;

    float minDistance = maxDistance;
    int   minIndex    = -1;
    float nextSampleDistance = 0;
    float dirx = rayDir.x;
    float diry = rayDir.y;
    float dirz = rayDir.z;

    float invDirx = rcp_safe(dirx);
    float invDiry = rcp_safe(diry);
    float invDirz = rcp_safe(dirz);
    int currentNode;
    
    int cacheSize = 0;
    Sample sampleCache[CACHE_MAX_SIZE];
    int todo[64]; //num of nodes to process
    int stackptr = 0;
    int barrier  = (int)END_FLAG;
    currentNode  = 0;

    todo[stackptr] = barrier;

    float ox = rayOrigin.x;
    float oy = rayOrigin.y;
    float oz = rayOrigin.z;
    float odirx = ox*invDirx;
    float odiry = oy*invDiry;
    float odirz = oz*invDirz;
    eavlVector4 color;
    color.x=0;
    color.y=0;
    color.z=0;
    color.w=0;

    while(currentNode!=END_FLAG) {
        


        if(currentNode>-1)
        {

            float4 n1=bvh.getValue(tet_bvh_in_tref, currentNode  ); //(txmin0, tymin0, tzmin0, txmax0)
            float4 n2=bvh.getValue(tet_bvh_in_tref, currentNode+1); //(tymax0, tzmax0, txmin1, tymin1)
            float4 n3=bvh.getValue(tet_bvh_in_tref, currentNode+2); //(tzmin1, txmax1, tymax1, tzmax1)
            
            float txmin0 =   n1.x*invDirx - odirx;       
            float tymin0 =   n1.y*invDiry - odiry;         
            float tzmin0 =   n1.z*invDirz - odirz;
            float txmax0 =   n1.w*invDirx - odirx;
            float tymax0 =   n2.x*invDiry - odiry;
            float tzmax0 =   n2.y*invDirz - odirz;
           
            float tmin0 = max(max(max(min(tymin0,tymax0),min(txmin0,txmax0)),min(tzmin0,tzmax0)),0.f); //maxDistance, how will this effect travseral if we change it on the fly
            float tmax0 = min(min(min(max(tymin0,tymax0),max(txmin0,txmax0)),max(tzmin0,tzmax0)), minDistance);
            
            bool traverseChild0 = (tmax0 >= tmin0); 

             
            float txmin1 =   n2.z*invDirx - odirx;       
            float tymin1 =   n2.w*invDiry - odiry;
            float tzmin1 =   n3.x*invDirz - odirz;
            float txmax1 =   n3.y*invDirx - odirx;
            float tymax1 =   n3.z*invDiry - odiry;
            float tzmax1 =   n3.w*invDirz - odirz;
            float tmin1 = max(max(max(min(tymin1,tymax1),min(txmin1,txmax1)),min(tzmin1,tzmax1)),0.f);
            float tmax1 = min(min(min(max(tymin1,tymax1),max(txmin1,txmax1)),max(tzmin1,tzmax1)), minDistance);
            
            bool traverseChild1 = (tmax1 >= tmin1);

        if(!traverseChild0 && !traverseChild1)
        {

            currentNode = todo[stackptr]; 
            stackptr--;
        }
        else
        {
            float4 n4 = bvh.getValue(tet_bvh_in_tref, currentNode+3); //(leftChild, rightChild, pad,pad)
            int leftChild  = (int)n4.x;
            int rightChild = (int)n4.y;

            currentNode = (traverseChild0) ? leftChild : rightChild;
            if(traverseChild1 && traverseChild0)
            {
                if(tmin0 > tmin1)
                {

                   
                    currentNode = rightChild;
                    stackptr++;
                    todo[stackptr] = leftChild;
                }
                else
                {   
                    stackptr++;
                    todo[stackptr] = rightChild;
                }

 
            }
        }
        }
        
        if(currentNode < 0 && currentNode != barrier)//check register usage
        {
            
            
            currentNode = -currentNode; //swap the neg address 
            int numTri = (int)tet_bvh_lf_raw.getValue(tet_bvh_lf_tref,currentNode)+1;
            int tetIndex=(int)tet_bvh_lf_raw.getValue(tet_bvh_lf_tref,currentNode+1); /*only one tet per leaf, we can get rid of the entire inner array */
            //cout<<"Checking primitive "<<tetIndex<<" "<< numTri<<endl;
            int hitCount = 0;
            float dist1 = 0;
            float dist2 = 0;
            float scalar1 = 0;
            float scalar2 = 0;
            for(int i=0;i<4;i++) /* Iterate over the triangles in the tetrahedron */
            {        
                     float4 a4 = verts.getValue(tet_verts_tref, (tetIndex*4+i%4)  ); /*Figure out a better way to do this*/
                     float4 b4 = verts.getValue(tet_verts_tref, (tetIndex*4+(1+i)%4));
                     float4 c4 = verts.getValue(tet_verts_tref, (tetIndex*4+(2+i)%4));
                     //cout<<(tetIndex*4+i%4  )<<" "<<(tetIndex*4+(1+i)%4)<<" "<<(tetIndex*4+(2+i)%4)<<endl;
                     //float4 d4 = verts.getValue(tet_verts_tref, ((tetIndex*4+3)+i)%4);
                     //cout<<a4.x<<" "<<a4.y<<" "<<a4.z<<endl;
                     //cout<<b4.x<<" "<<b4.y<<" "<<b4.z<<endl;
                     //cout<<c4.x<<" "<<c4.y<<" "<<c4.z<<endl;
                    eavlVector3 e1( b4.x-a4.x , b4.y-a4.y, b4.z-a4.z );
                    eavlVector3 e2( c4.x-a4.x , c4.y-a4.y, c4.z-a4.z ); 
                    


                    eavlVector3 p;
                    p.x = diry*e2.z - dirz*e2.y;
                    p.y = dirz*e2.x - dirx*e2.z;
                    p.z = dirx*e2.y - diry*e2.x;
                    float dot = e1*p;
                    if(dot != 0.f)
                    {   //cout<<" dot ";
                        dot = 1.f/dot;
                        eavlVector3 t;
                        t.x = ox - a4.x;
                        t.y = oy - a4.y;
                        t.z = oz - a4.z;

                        float u = (t*p)*dot;
                        if(u >= 0.f && u <= 1.f)
                        {//cout<<" u ";
                            eavlVector3 q = t%e1;
                            float v = (dirx*q.x + diry*q.y + dirz*q.z)*dot;
                            if(v >= 0.f && v <= 1.f)  //hits 3 or 1 
                            {//cout<<" v ";
                                float dist = (e2*q)*dot;
                                //if((dist > EPSILON && dist < minDistance) && !(u+v>1) )
                                if((dist < minDistance) && !(u+v>1) )
                                {
                                    float scalar = a4.w*u + b4.w*v + c4.w*(1 - u - v); //lerp
                                    //scalar =.5f;
                                    hitCount++;
                                    
                                    if(hitCount == 1)
                                    {
                                      dist1 = dist; //we are looking for two distances  
                                      scalar1 =scalar;
                                    } 
                                    else
                                    {
                                      dist2 = dist;
                                      scalar2 = scalar;  
                                    } 
                                    //minDistance = dist;
                                    //minIndex = triIndex;
                                    //cout<<"Hit t "<<i<<" at tet "<<tetIndex<<endl;
                
                                }
                            }
                        }

                    }
                   
            }
            /* now see if the sample point in within this range */
            bool gotSample = false;
            if(hitCount == 2) /*not sure what to so about degenerates*/
            {   if(dist1 > dist2)
                {
                    float t = dist1;
                    dist1 = dist2;
                    dist2 = t;
                    t = scalar1;
                    scalar2 = scalar1;
                    scalar1 = t;
                    if(nextSampleDistance == 0) { nextSampleDistance = dist1; } //??????
                } 

                if(cacheSize > 0)
                {
                    bool entryFound = true;
                    while(entryFound)
                    {   
                        entryFound = false;
                        for(int j=0; j< CACHE_MAX_SIZE; j++)
                        {
                            //cout<<"Searching cache "<<j<<" isOpen "<<sampleCache[j].open<<endl;
                            if(sampleCache[j].open == 1)
                            {
                                if(sampleCache[j].d2<nextSampleDistance) {sampleCache[j].open = 0; cacheSize--;}
                                gotSample =false;
                                while(sampleCache[j].d1<=nextSampleDistance && sampleCache[j].d2>=nextSampleDistance)
                                {
                                    cout<<"######### CACHE SAMPLE ###########  "<<nextSampleDistance<<endl;
                                    //cout<<sampleCache[j].d1<<" "<<sampleCache[j].d2<<" "<<nextSampleDistance<<endl;
                                    float s = lerp(sampleCache[j].s1,sampleCache[j].s2, clamp((nextSampleDistance - sampleCache[j].s1) / (sampleCache[j].s2 - sampleCache[j].s1), 0.0f, 1.0f));
                                    cout<<"S "<<s<<endl;  
                                    int   colorIdx = floor(s*cmapSize);
                                    float4 c = cmap.getValue(color_map_tref, colorIdx); //divide by number of samples
                                    return eavlVector4(c.x,c.y,c.z,c.w);
                                    color.x += c.x * (1.-color.w)*c.w;
                                    color.y += c.y * (1.-color.w)*c.w;
                                    color.z += c.z * (1.-color.w)*c.w;
                                    color.w += c.w * (1.-color.w)*c.w;
                                    nextSampleDistance += sampleDelta;
                                    gotSample = true;
                                }
                                if(gotSample)
                               { 
                                   sampleCache[j].open = 0;
                                   entryFound = true; // keep scanning for another entry
                                   cacheSize--;
                               }
                            }
                        }
                    }
                }
                //cout<<"Current Node "<<currentNode<<endl;
                //cout<<"Node Range: "<<dist1<<" - "<<dist2<<" Looking for "<<nextSampleDistance<< endl;
                if(dist1 <= nextSampleDistance && nextSampleDistance <= dist2)
                {
                    while(dist1 <= nextSampleDistance && nextSampleDistance <= dist2)
                    {
                        cout<<"######### SAMPLE ###########   "<<nextSampleDistance<<endl;
                        float s = lerp(scalar1,scalar2, clamp((nextSampleDistance - scalar1) / (scalar2 - scalar1), 0.0f, 1.0f)); 
                        cout<<"S "<<s<<endl;             
                        int   colorIdx = floor(s*cmapSize);
                        float4 c = cmap.getValue(color_map_tref, colorIdx); //divide by number of samples
                        return eavlVector4(c.x,c.y,c.z,c.w);
                        cout<<"Color "<<c.x<<" "<<c.y<<" "<<c.z<<" "<<c.w<<endl;
                        color.x += c.x * (1.-color.w)*c.w;
                        color.y += c.y * (1.-color.w)*c.w;
                        color.z += c.z * (1.-color.w)*c.w;
                        color.w += c.w * (1.-color.w)*c.w;
                         cout<<"Color Acc "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<endl;
                        nextSampleDistance += sampleDelta;
                    }
                }
                else
                {
                    if(dist1 > nextSampleDistance) //cahce future entry range
                    {
                        for(int j=0; j<= CACHE_MAX_SIZE; j++)
                        {
                            if(j == CACHE_MAX_SIZE) {cout<<"@@@@@@ Cache blown @@@@@@@@@"<<endl; break;}
                            if(sampleCache[j].open == 0)
                            {   //cout<<"Caching sample : "<<dist1<<" - "<<dist2<<" "<<j<<endl;
                                sampleCache[j].d1 = dist1;
                                sampleCache[j].d2 = dist2;
                                sampleCache[j].s1 = scalar1;
                                sampleCache[j].s2 = scalar2;
                                sampleCache[j].open = 1;
                                cacheSize++;
                                break;
                            }
                        }


                    }
                }
            }
            
            
            
           

                currentNode=todo[stackptr];
                stackptr--;
            
            
            
            
        }

    }
 distance=minDistance;
 //f(color.x!=0)
 //{
 //   color.x = 1;
 //   color.y = 1;
  //  color.z = 0;
  //  color.w = 1;
 //}
 return color;
}



struct RayIntersectFunctor{


    eavlConstArrayV2<float4> verts;
    eavlConstArrayV2<float4> cmap;
    eavlConstArrayV2<float4> bvh;
    eavlConstArrayV2<float>  bvh_inner;
    primitive_t              primitiveType;
    float                    sampleDelta;
    int                      colorMapSize;

    RayIntersectFunctor(eavlConstArrayV2<float4> *_verts, eavlConstArrayV2<float4> *theBvh,eavlConstArrayV2<float> *_bvh_inner, primitive_t _primitveType, float _sampleDelta,int _colorMapSize,eavlConstArrayV2<float4> *_cmap)
        :verts(*_verts),
         bvh(*theBvh),
         bvh_inner(*_bvh_inner),
         primitiveType(_primitveType),
         sampleDelta(_sampleDelta),
         colorMapSize(_colorMapSize),
         cmap(*_cmap)
    {}                                                 
    EAVL_HOSTDEVICE tuple<float,float,float,float> operator()( tuple<float,float,float,float,float,float> rayTuple){

        int   minHit = -1; 
        float distance;
        eavlVector3 rayOrigin(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        eavlVector3       ray(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector4 c;
        if(primitiveType == TET)
        {
            c = getIntersectionTet(ray, rayOrigin,bvh,bvh_inner, verts,INFINITE,distance,sampleDelta, cmap, colorMapSize);
        } 
        
        
        return tuple<float,float,float,float>(c.x,c.y,c.z,c.w);
    }
};


void eavlVolumeRendererMutator::setColorMap3f(float* cmap,int size)
{
    colorMapSize = size;
    if(color_map_array != NULL)
    {
        color_map_array->unbind(color_map_tref);
        delete color_map_array;
    }
    if(color_map_raw!=NULL)
    {
        delete color_map_raw;
    }
    color_map_raw= new float[size*4];
    
    for(int i=0;i<size;i++)
    {
        color_map_raw[i*4  ] = cmap[i*3  ];
        color_map_raw[i*4+1] = cmap[i*3+1];
        color_map_raw[i*4+2] = cmap[i*3+2];
        color_map_raw[i*4+3] = .05;          //test Alpha
        //cout<<cmap[i*3]<<" "<<cmap[i*3+1]<<" "<<cmap[i*3+2]<<endl;
    }
    color_map_array = new eavlConstArrayV2<float4>((float4*)color_map_raw, colorMapSize, color_map_tref);
}

void eavlVolumeRendererMutator::setDefaultColorMap()
{   cout<<"setting defaul color map"<<endl;
    if(color_map_array!=NULL)
    {
        color_map_array->unbind(color_map_tref);
        delete color_map_array;
    }
    if(color_map_raw!=NULL)
    {
        delete[] color_map_raw;
    }
    //two values all 1s
    colorMapSize=2;
    color_map_raw= new float[8];
    for(int i=0;i<8;i++) color_map_raw[i]=1.f;
    color_map_array = new eavlConstArrayV2<float4>((float4*)color_map_raw, colorMapSize, color_map_tref);
    cout<<"Done setting defaul color map"<<endl;

}

void eavlVolumeRendererMutator::clearFrameBuffer(eavlFloatArray *r,eavlFloatArray *g,eavlFloatArray *b,eavlFloatArray *a)
{
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(r),
                                            eavlOpArgs(r),
                                            FloatMemsetFunctor(0)),
                                            "memset");
    eavlExecutor::Go();
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(g),
                                            eavlOpArgs(g),
                                            FloatMemsetFunctor(0)),
                                            "memset");
    eavlExecutor::Go();
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(b),
                                            eavlOpArgs(b),
                                            FloatMemsetFunctor(0)),
                                            "memset");
    eavlExecutor::Go();
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(a),
                                            eavlOpArgs(a),
                                            FloatMemsetFunctor(0)),
                                            "memset");
    eavlExecutor::Go();

}

void eavlVolumeRendererMutator::allocateArrays()
{
	deleteClassPtr(rayDirX);
    deleteClassPtr(rayDirY);
    deleteClassPtr(rayDirZ);

    deleteClassPtr(rayOriginX);
    deleteClassPtr(rayOriginY);
    deleteClassPtr(rayOriginZ);

    deleteClassPtr(r);
    deleteClassPtr(g);
    deleteClassPtr(b);
    deleteClassPtr(a);
    deleteClassPtr(frameBuffer);
    deleteClassPtr(indexes);
    deleteClassPtr(mortonIndexes);
    deleteClassPtr(tempFloat);

    indexes          = new eavlIntArray("indexes",1,size);
    mortonIndexes    = new eavlIntArray("mortonIdxs",1,size);

    rayDirX          = new eavlFloatArray("x",1,size);
    rayDirY          = new eavlFloatArray("y",1,size);
    rayDirZ          = new eavlFloatArray("z",1,size);

    rayOriginX       = new eavlFloatArray("x",1,size);
    rayOriginY       = new eavlFloatArray("y",1,size);
    rayOriginZ       = new eavlFloatArray("z",1,size);

    r                = new eavlFloatArray("r",1,size);
    g                = new eavlFloatArray("b",1,size);
    b                = new eavlFloatArray("g",1,size);
    a                = new eavlFloatArray("g",1,size);
    tempFloat        = new eavlFloatArray("g",1,size);

    frameBuffer      = new eavlFloatArray("",1, width*height*4);
    sizeDirty = false;
}

void eavlVolumeRendererMutator::init()
{   cout<<"Init"<<endl;
	size = height*width;
	if(sizeDirty) 
    {
        allocateArrays();
        createRays(); //creates the morton ray indexes
    }

    /* Set ray origins to the eye */
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(indexes), //dummy arg
                                             eavlOpArgs(rayOriginX,rayOriginY,rayOriginZ),
                                             FloatMemsetFunctor3to3(camera.position.x,camera.position.y,camera.position.z)),
                                             "init");
    eavlExecutor::Go();

    /* Copy morton indexes into idxs*/ 
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(mortonIndexes),
                                             eavlOpArgs(indexes),
                                             FloatMemcpyFunctor1to1()),
                                             "cpy");
    eavlExecutor::Go();

    if(geomDirty) extractGeometry();
}

void eavlVolumeRendererMutator::extractGeometry()
{
    if(verbose) cerr<<"Extracting Geometry"<<endl;
    freeRaw();
    freeTextures();
    //nunTets = scene.getNumTets();
    tet_verts_raw = scene.getTetPtr();
    int tet_bvh_in_size = 0;
    int tet_bvh_lf_size = 0;
    cout<<"Building BVH...."<<endl;
    SplitBVH *testSplit= new SplitBVH(tet_verts_raw, numTets, TET); // 0=triangle
    cout<<"Done building."<<endl;
    testSplit->getFlatArray(tet_bvh_in_size, tet_bvh_lf_size, tet_bvh_in_raw, tet_bvh_lf_raw);
    //if( writeCache) writeBVHCache(tri_bvh_in_raw, tri_bvh_in_size, tri_bvh_lf_raw, tri_bvh_lf_size, bvhCacheName.c_str());
    delete testSplit;

    tet_bvh_in_array   = new eavlConstArrayV2<float4>( (float4*)tet_bvh_in_raw, tet_bvh_in_size/4, tet_bvh_in_tref);
    tet_bvh_lf_array   = new eavlConstArrayV2<float>( tet_bvh_lf_raw, tet_bvh_lf_size, tet_bvh_lf_tref);
    tet_verts_array    = new eavlConstArrayV2<float4>( (float4*) tet_verts_raw, numTets*4, tet_verts_tref);

    geomDirty=false;
}



void eavlVolumeRendererMutator::Execute()
{
    numTets = scene.getNumTets();
    if(numTets == 0 ) 
    {  
        cout<<"No primitives to render. "<<endl;
        return;
    }
    
    init();
    clearFrameBuffer(r,g,b,a);
    camera.look = camera.lookat - camera.position;
    //init camera rays
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(indexes),
                                             eavlOpArgs(rayDirX ,rayDirY, rayDirZ),
                                             RayGenFunctor(width, height, camera.fovx, camera.fovy, camera.look, camera.up, camera.zoom)),
                                             "ray gen");
    eavlExecutor::Go();

    int ttraverse;
    if(verbose) ttraverse = eavlTimer::Start();
    //eavlExecutor::SetExecutionMode(eavlExecutor::ForceCPU);
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rayDirX,rayDirY,rayDirZ,rayOriginX,rayOriginY,rayOriginZ),
                                             eavlOpArgs(r,g,b,a),
                                             RayIntersectFunctor(tet_verts_array,tet_bvh_in_array,tet_bvh_lf_array,TET,sampleDelta, colorMapSize, color_map_array)),
                                                                                                        "intersect");
    eavlExecutor::Go();
    //eavlExecutor::SetExecutionMode(eavlExecutor::ForceGPU);

    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(r, g, b,a),
                                                 eavlOpArgs(eavlIndexable<eavlFloatArray>(frameBuffer,*redIndexer),
                                                            eavlIndexable<eavlFloatArray>(frameBuffer,*greenIndexer),
                                                            eavlIndexable<eavlFloatArray>(frameBuffer,*blueIndexer),
                                                            eavlIndexable<eavlFloatArray>(frameBuffer,*alphaIndexer)),
                                                 FloatMemcpyFunctor4to4()),
                                                 "memcopy");
     eavlExecutor::Go();
    if(verbose) cout<<"Traversal   RUNTIME: "<<eavlTimer::Stop(ttraverse,"traverse")<<endl;

}

void eavlVolumeRendererMutator::createRays()
{
    float fwidth=(float)width;
    float fheight=(float)height;
    float  w,h;

    raySort *rayArray= new raySort[size]; // since this is happening every frame it should not be allocated and deleted constantly.
                                          
    for(int i=0; i<size;i++)
    {
        rayArray[i].id=i;
        w = (float)(i%width)/fwidth;
        h = (float) (i/width)/fheight;
        rayArray[i].mortonCode=morton2D(w,h);
    }
    //std::sort(rayArray,rayArray+size,spacialCompare);
    cout<<endl;
    for(int i=0; i<size;i++)
    {
        mortonIndexes->SetValue(i, rayArray[i].id);
    }
    delete[] rayArray; 
} 


void eavlVolumeRendererMutator::freeRaw()
{
    
    deleteArrayPtr(tet_verts_raw);
    deleteArrayPtr(tet_bvh_in_raw);
    deleteArrayPtr(tet_bvh_lf_raw);
    cout<<"Free raw"<<endl;

}


void eavlVolumeRendererMutator::freeTextures()
{
    cout<<"Free textures"<<endl;
   if (tet_bvh_in_array != NULL) 
    {
        tet_bvh_in_array->unbind(tet_bvh_in_tref);
        delete tet_bvh_in_array;
        tet_bvh_in_array = NULL;
    }
    if (tet_bvh_lf_array != NULL) 
    {
        tet_bvh_lf_array->unbind(tet_bvh_lf_tref);
        delete tet_bvh_lf_array;
        tet_bvh_lf_array = NULL;
    }
    if (tet_verts_array != NULL) 
    {
        tet_verts_array ->unbind(tet_verts_tref);
        delete tet_verts_array;
        tet_verts_array = NULL;
    }
}