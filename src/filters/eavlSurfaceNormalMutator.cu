#include "hip/hip_runtime.h"
// Copyright 2010-2012 UT-Battelle, LLC.  See LICENSE.txt for more information.
#include "eavlSurfaceNormalMutator.h"

#include "eavl.h"
#include "eavlFilter.h"
#include "eavlCellSet.h"
#include "eavlField.h"
#include "eavlDataSet.h"
#include "eavlTopologyMapOp_3_0_3.h"
#include "eavlField.h"
#include "eavlVector3.h"
#include "eavlException.h"
#include "eavlExecutor.h"

class FaceNormalFunctor
{
  public:
    EAVL_FUNCTOR void operator()(int shapeType, int n,
                                 float x[], float y[], float z[],
                                 float &ox, float &oy, float &oz)
    {
        // should we treat EAVL_PIXEL differently here?
        float p1x = x[1] - x[0];
        float p1y = y[1] - y[0];
        float p1z = z[1] - z[0];
	    
        float p2x = x[2] - x[1];
        float p2y = y[2] - y[1];
        float p2z = z[2] - z[1];
	    
        ox = p1y*p2z - p1z*p2y;
        oy = p1z*p2x - p1x*p2z;
        oz = p1x*p2y - p1y*p2x;
        float len = sqrt(ox*ox+oy*oy+oz*oz);
        if (len>0)
        {
            ox /= len;
            oy /= len;
            oz /= len;
        }
    }
};

eavlSurfaceNormalMutator::eavlSurfaceNormalMutator()
{
}


void
eavlSurfaceNormalMutator::Execute()
{
    int inCellSetIndex = dataset->GetCellSetIndex(cellsetname);
    eavlCellSet *inCells = dataset->GetCellSet(cellsetname);

    // input arrays are from the coordinates
    eavlCoordinates *cs = dataset->coordinateSystems[0];
    if (cs->GetDimension() != 3)
        THROW(eavlException,"eavlNodeToCellOp assumes 3D coordinates");

    eavlCoordinateAxisField *axis0 = dynamic_cast<eavlCoordinateAxisField*>(cs->GetAxis(0));
    eavlCoordinateAxisField *axis1 = dynamic_cast<eavlCoordinateAxisField*>(cs->GetAxis(1));
    eavlCoordinateAxisField *axis2 = dynamic_cast<eavlCoordinateAxisField*>(cs->GetAxis(2));

    if (!axis0 || !axis1 || !axis2)
        THROW(eavlException,"eavlNodeToCellOp expects only field-based coordinate axes");

    eavlField *field0 = dataset->GetField(axis0->GetFieldName());
    eavlField *field1 = dataset->GetField(axis1->GetFieldName());
    eavlField *field2 = dataset->GetField(axis2->GetFieldName());
    eavlArray *arr0 = field0->GetArray();
    eavlArray *arr1 = field1->GetArray();
    eavlArray *arr2 = field2->GetArray();
    if (!arr0 || !arr1 || !arr2)
    {
        THROW(eavlException,"eavlNodeToCellOp assumes single-precision float arrays");
    }

    eavlArrayWithLinearIndex i0(arr0, axis0->GetComponent());
    eavlArrayWithLinearIndex i1(arr1, axis1->GetComponent());
    eavlArrayWithLinearIndex i2(arr2, axis2->GetComponent());
    if (field0->GetAssociation() == eavlField::ASSOC_WHOLEMESH)
        i0.mul = 0;
    if (field1->GetAssociation() == eavlField::ASSOC_WHOLEMESH)
        i1.mul = 0;
    if (field2->GetAssociation() == eavlField::ASSOC_WHOLEMESH)
        i2.mul = 0;
    
    eavlLogicalStructureRegular *logReg = dynamic_cast<eavlLogicalStructureRegular*>(dataset->logicalStructure);
    if (logReg)
    {
        eavlRegularStructure &reg = logReg->GetRegularStructure();

        if (field0->GetAssociation() == eavlField::ASSOC_LOGICALDIM)
            i0 = eavlArrayWithLinearIndex(arr0, axis0->GetComponent(), reg, field0->GetAssocLogicalDim());
        if (field1->GetAssociation() == eavlField::ASSOC_LOGICALDIM)
            i1 = eavlArrayWithLinearIndex(arr1, axis1->GetComponent(), reg, field1->GetAssocLogicalDim());
        if (field2->GetAssociation() == eavlField::ASSOC_LOGICALDIM)
            i2 = eavlArrayWithLinearIndex(arr2, axis2->GetComponent(), reg, field2->GetAssocLogicalDim());
    }

    eavlFloatArray *out = new eavlFloatArray("surface_normals", 3,
                                             inCells->GetNumCells());

    eavlExecutor::AddOperation(new eavlTopologyMapOp_3_0_3<FaceNormalFunctor>(
                                      inCells, EAVL_NODES_OF_CELLS,
                                      i0, i1, i2,
                                      eavlArrayWithLinearIndex(out, 0),
                                      eavlArrayWithLinearIndex(out, 1),
                                      eavlArrayWithLinearIndex(out, 2),
                                      FaceNormalFunctor()),
                               "surface normal");
    eavlExecutor::Go();

    eavlField *cellnormalfield = new eavlField(0, out,
                                               eavlField::ASSOC_CELL_SET,
                                               inCellSetIndex);
    dataset->fields.push_back(cellnormalfield);
};
